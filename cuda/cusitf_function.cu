#include "hip/hip_runtime.h"
#include "cusitf_function.h"

#define CHECK(res) { if(res != hipSuccess){printf("Error ：%s:%d , ", __FILE__,__LINE__);   \
printf("code : %d , reason : %s \n", res,hipGetErrorString(res));exit(-1);}}


__global__ void foo()
{
    printf("CUDA!\n");
}


void useCUDA()
{

    foo<<<1,5>>>();
    CHECK(hipDeviceSynchronize());

}
void cuGaussianBlur(InputArray src, OutputArray dst, Size ksize,
                    double sigmaX, double sigmaY = 0,
                    int borderType = BORDER_DEFAULT )
{

}
