#include "hip/hip_runtime.h"
#include "cusitf_function_H.h"


#define MESSAGE 1


#define __MAXSIZECON 64*2+1
__constant__ float coeffGaussKernel[__MAXSIZECON];
texture<float, 1, hipReadModeElementType> texRef;

/***********
//This is an adjustable option which control the gaussKernel size. \
//when the kernel size less than 32*2+1 or kernel radius less than 32,the ROW_HALO_STEP set 1 \
//and the COLUMNS_HALO_STEPS set 2 will has a good performance.But when the kernel size is more \
//than 32 less than 64, the ROW_HALO_STEP should set 2 and the COLUMNS_HALO_STEPS should set 4
//The ROW_HALO_STEP will set 2 and the COLUMNS_HALO_STEPS will set 4 by default
***********/

///////////////////////
/// \brief GaussianBlurKernelRow
/// \param d_data
/// \param out
/// \param w
/// \param h
/// \param ksize
/// \param pitch
/// Only support the kernel size less than 32*2+1(ROW_HALO_STEP*ROW_BLOCK_DIM_X(32) is the radius)
/// Reference the cuda-sample 'convolutionSeparable'.
/// The boundary is set 0.
/// If adjust the ROW_HALO_STEP 2,that is ok.
//////////////////////

#define   ROW_BLOCK_DIM_X 32
#define   ROW_BLOCK_DIM_Y 8
#define  ROW_UNROLL_STEPS 4
#define     ROW_HALO_STEP 2

__global__ void GaussianBlurKernelRow(
    float *d_data,
    float *out,
    int w,
    int h,
    int ksize,
    int pitch
)
{

    __shared__ float s[ROW_BLOCK_DIM_Y][ROW_BLOCK_DIM_X*(ROW_UNROLL_STEPS+ROW_HALO_STEP*2)];

    //base shared memory coordinate
    int baseX = (blockIdx.x*ROW_UNROLL_STEPS-ROW_HALO_STEP)*blockDim.x + threadIdx.x;
    int baseY = blockIdx.y*blockDim.y+threadIdx.y;

    //the data basing shared memory coordinate
    d_data += baseY * pitch + baseX;
    out    += baseY * pitch + baseX;

    //Load main data
#pragma unroll
    for(int i = ROW_HALO_STEP;i<ROW_UNROLL_STEPS+ROW_HALO_STEP;i++)
        //s[threadIdx.y][threadIdx.x+ i * ROW_BLOCK_DIM_X] = d_data[ROW_BLOCK_DIM_X * i];
        s[threadIdx.y][threadIdx.x+ i * ROW_BLOCK_DIM_X] = (baseX + ROW_BLOCK_DIM_X * i < w ) ? d_data[ROW_BLOCK_DIM_X * i] : 0;


    //Load left halo
    //left halo exist when this is threads in the imgae patch.
#pragma unroll
    for (int i = 0; i < ROW_HALO_STEP; i++)
    {
        s[threadIdx.y][threadIdx.x + i * ROW_BLOCK_DIM_X] = (baseX >= -i * ROW_BLOCK_DIM_X ) ? d_data[i * ROW_BLOCK_DIM_X] : 0;
    }


    //Load right halo
    //left halo exist when this is threads in the imgae patch.
#pragma unroll
    for (int i = ROW_HALO_STEP + ROW_UNROLL_STEPS; i < ROW_HALO_STEP + ROW_UNROLL_STEPS + ROW_HALO_STEP; i++)
    {
        s[threadIdx.y][threadIdx.x + i * ROW_BLOCK_DIM_X] = (w - baseX > i * ROW_BLOCK_DIM_X) ? d_data[i * ROW_BLOCK_DIM_X] : 0;
    }

    __syncthreads();


    int b = (ksize -1) /2;
    for (int i = ROW_HALO_STEP; i < ROW_HALO_STEP + ROW_UNROLL_STEPS; i++)
    {
        float sum = 0;

#pragma unroll
        for (int j = -b; j <= b; j++)
        {
            sum += coeffGaussKernel[b-j] * s[threadIdx.y][threadIdx.x + i * ROW_BLOCK_DIM_X + j];
        }

        out[i * ROW_BLOCK_DIM_X] = sum;
    }

    //old version
//     int b = (ksize -1) /2;
//     if(x>=b && x<w-b && y>=0 && y<h){
//        #pragma unroll
//        float sum = 0;
//        for(int i = -b;i<=b;i++){
//            sum += d_data[y*pitch+x+i]*coeffGaussKernel[i+b];
//        }
//        out[y*pitch+x] = sum;
//     }
}
///////////////////////////////////
/// \brief GaussianBlurKernelCol
/// \param d_data
/// \param out
/// \param w
/// \param h
/// \param ksize
/// \param pitch
/// There is a different with row that the col has not the pitch which could make sure the \
/// all thereds in image aera.
/// Reference the cuda-sample 'convolutionSeparable'
/// The boundary is set 0.
/// The minimum y size is 64(COLUMNS_BLOCKDIM_Y*COLUMNS_RESULT_STEPS)
//////////////////////////////////

#define   COLUMNS_BLOCKDIM_X 32
#define   COLUMNS_BLOCKDIM_Y 16
#define COLUMNS_RESULT_STEPS 4
#define   COLUMNS_HALO_STEPS 4

__global__ void GaussianBlurKernelCol(
    float *d_data,
    float *out,
    int w,
    int h,
    int ksize,
    int pitch
)
{

    __shared__ float s_Data[COLUMNS_BLOCKDIM_X][(COLUMNS_RESULT_STEPS + 2 * COLUMNS_HALO_STEPS) * COLUMNS_BLOCKDIM_Y + 1];

    //Offset to the upper halo edge
    const int baseX = blockIdx.x * COLUMNS_BLOCKDIM_X + threadIdx.x;
    const int baseY = (blockIdx.y * COLUMNS_RESULT_STEPS - COLUMNS_HALO_STEPS) * COLUMNS_BLOCKDIM_Y + threadIdx.y;
    d_data += baseY * pitch + baseX;
    out    += baseY * pitch + baseX;
    int b = (ksize -1) /2;

    //fill the shared memory not consider the upper halo,so it limit the minimum y size is 64(COLUMNS_BLOCKDIM_Y*COLUMNS_RESULT_STEPS)
    if(baseY + (COLUMNS_RESULT_STEPS+COLUMNS_HALO_STEPS)*COLUMNS_BLOCKDIM_Y >= h && baseY + COLUMNS_HALO_STEPS*COLUMNS_BLOCKDIM_Y < h)
    {


        //Main data and lower halo
#pragma unroll
        for (int i = COLUMNS_HALO_STEPS; i <  COLUMNS_RESULT_STEPS + COLUMNS_HALO_STEPS*2 ; i++)
        {
            s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = (baseY + i * COLUMNS_BLOCKDIM_Y < h) ? d_data[i * COLUMNS_BLOCKDIM_Y * pitch] : 0;
        }

        //Upper halo
#pragma unroll

        for (int i = 0; i < COLUMNS_HALO_STEPS; i++)
        {
            s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = (baseY >= -i * COLUMNS_BLOCKDIM_Y) ? d_data[i * COLUMNS_BLOCKDIM_Y * pitch] : 0;
        }


        for (int i = COLUMNS_HALO_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i++)
        {
            float sum = 0;
#pragma unroll

            for (int j = -b ; j <= b; j++)
            {
                sum += coeffGaussKernel[b - j]* s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y + j];
            }

            if(baseY + i * COLUMNS_BLOCKDIM_Y < h) {
                out[i * COLUMNS_BLOCKDIM_Y * pitch] = sum;
            }
        }

        return;
    }



    //Main data
#pragma unroll

    for (int i = COLUMNS_HALO_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i++)
    {
        s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = d_data[i * COLUMNS_BLOCKDIM_Y * pitch];
    }

    //Upper halo
#pragma unroll

    for (int i = 0; i < COLUMNS_HALO_STEPS; i++)
    {
        s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = (baseY >= -i * COLUMNS_BLOCKDIM_Y) ? d_data[i * COLUMNS_BLOCKDIM_Y * pitch] : 0;
    }

    //Lower halo
#pragma unroll

    for (int i = COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS + COLUMNS_HALO_STEPS; i++)
    {
        s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y]= (h - baseY > i * COLUMNS_BLOCKDIM_Y) ? d_data[i * COLUMNS_BLOCKDIM_Y * pitch] : 0;
    }

    //Compute and store results
    __syncthreads();


#pragma unroll

    for (int i = COLUMNS_HALO_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i++)
    {
        float sum = 0;
#pragma unroll

        for (int j = -b ; j <= b; j++)
        {
            sum += coeffGaussKernel[b - j]* s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y + j];
        }

        out[i * COLUMNS_BLOCKDIM_Y * pitch] = sum;
    }

#if 0
    if(y>=b && y<h-b && x>=0 && x<w){
        #pragma unroll
        for(int i = 0;i<ksize;i++){
            if(i<b){
                out[y*pitch+x] += d_data[(y-b+i)*pitch+x]*coeffGaussKernel[i];
            }
            else{
                out[y*pitch+x] += d_data[(y+i-b)*pitch+x]*coeffGaussKernel[i];
            }
        }
    }
#else
//    if(y>=b && y<h-b && x>=0 && x<w){
//       #pragma unroll
//       float sum = 0;
//       for(int i = -b;i<=b;i++){
//           sum += d_data[(y+i)*pitch+x]*coeffGaussKernel[i+b];
//       }
//       out[y*pitch+x] = sum;
//    }
#endif

}

__global__ void GaussianBlurKernelRTex(float *out,int w,int h,int ksize)
{

    int x = blockIdx.x*blockDim.x+threadIdx.x;
    int y = blockIdx.y*blockDim.y+threadIdx.y;

    int b = (ksize -1) /2;

    if(x>=b && x<w-b && y>=0 && y<h){
        #pragma unroll
        for(int i = 0;i<ksize;i++){
            if(i<b){
                out[y*w+x] += tex1Dfetch(texRef,y*w+x-b+i)*coeffGaussKernel[i];
            }
            else{
                out[y*w+x] += tex1Dfetch(texRef,y*w+x+i-b)*coeffGaussKernel[i];
            }
        }
    }

}


void cuGaussianBlur(CudaImage &cuImg,float sigma)
{
    //ksize.width = cvRound(sigma*(depth == CV_8U ? 3 : 4)*2 + 1)|1;
    //createsize
    //getkernel

    assert(sigma>0);
    assert(1);
    int kernelSize = 0;

    //sigma = sqrtf(sigma * sigma - 0.5 * 0.5 * 4);

    kernelSize = cvRound(sigma*4*2 + 1)|1;

    Mat kx;
    kx = getGaussianKernel(kernelSize,sigma,CV_32F);

    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(coeffGaussKernel),(float*)kx.data,sizeof(float)*kernelSize));

    dim3 BlockRow(ROW_BLOCK_DIM_X,ROW_BLOCK_DIM_Y);
    dim3 GridRow(iDivUp(cuImg.pitch,BlockRow.x*ROW_UNROLL_STEPS),iDivUp(cuImg.height,BlockRow.y));

    float *tmp_data,*tmp_data1;

    size_t pitch;
    // safeCall(hipMalloc(&tmp_data,cuImg.width*cuImg.height*sizeof(float)));
    safeCall(hipMallocPitch((void**)&tmp_data, (size_t*) &pitch, (size_t) cuImg.width*sizeof(float),  (size_t) cuImg.height));

    GaussianBlurKernelRow<<<GridRow,BlockRow>>>(cuImg.d_data,tmp_data,cuImg.width,cuImg.height,kernelSize,cuImg.pitch);
    safeCall(hipDeviceSynchronize());

    safeCall(hipMallocPitch((void**)&tmp_data1, (size_t*) &pitch, (size_t) cuImg.width*sizeof(float),  (size_t) cuImg.height));

    dim3 BlockCol(COLUMNS_BLOCKDIM_X,COLUMNS_BLOCKDIM_Y);
    dim3 GridCol(iDivUp(cuImg.pitch,BlockCol.x),iDivUp(cuImg.height,BlockCol.y*COLUMNS_RESULT_STEPS));


    //safeCall(hipMalloc(&tmp_data1,cuImg.width*cuImg.height*sizeof(float)));
    GaussianBlurKernelCol<<<GridCol,BlockCol>>>(tmp_data,tmp_data1,cuImg.width,cuImg.height,kernelSize,cuImg.pitch);
    safeCall(hipDeviceSynchronize());

    /*device data has not copy to host yet*/



#if 1
    Mat dis(cuImg.height,cuImg.width,CV_32F);
    //safeCall(hipMemcpy(dis.data,tmp_data1,cuImg.width*cuImg.height*sizeof(float),hipMemcpyDeviceToHost));
    safeCall(hipMemcpy2D(dis.data,cuImg.width*sizeof(float),tmp_data1,cuImg.pitch*sizeof(float),cuImg.width*sizeof(float),(size_t) cuImg.height,hipMemcpyDeviceToHost));

//    Mat dis(cuImg.height,cuImg.pitch,CV_32F);
//    //safeCall(hipMemcpy(dis.data,tmp_data1,cuImg.width*cuImg.height*sizeof(float),hipMemcpyDeviceToHost));
//    safeCall(hipMemcpy2D(dis.data,cuImg.pitch*sizeof(float),tmp_data1,cuImg.pitch*sizeof(float),cuImg.width*sizeof(float),(size_t) cuImg.height,hipMemcpyDeviceToHost));


    //    for(int i = 0;i<dis.rows;i++)
//    {
//        float *p = dis.ptr<float>(i);
//        for(int j = 0;j<dis.cols;j++){
//            p[j] = cuImg.h_data[i*dis.cols+j];
//            //std::cout<<p[j]<<" ";
//        }
//        //std::cout<<std::endl;
//    }
//    memcpy(dis.data,tmp_data1,cuImg.width*cuImg.height*sizeof(float));
    Mat gray;
    dis.convertTo(gray,DataType<uchar>::type, 1, 0);

    cvNamedWindow("ss",CV_WINDOW_NORMAL);
    imshow("ss",gray);
    waitKey();
#endif

    hipFree(tmp_data);
    hipFree(tmp_data1);

#if MESSAGE == 0
    std::cout<<kernelSize<<std::endl;
    for(int i= 0 ;i<kx.rows;i++)
        for(int j = 0;j<kx.cols;j++){
            std::cout<<kx.at<float>(i,j)<<std::endl;
        }
#endif


}






void disMatf(CudaImage &cuImg){
    Mat dis(cuImg.height,cuImg.width,CV_32F);

    for(int i = 0;i<dis.rows;i++)
    {
        float *p = dis.ptr<float>(i);
        for(int j = 0;j<dis.cols;j++){
            p[j] = cuImg.h_data[i*dis.cols+j];
            //std::cout<<p[j]<<" ";
        }
        //std::cout<<std::endl;
    }
    //memcpy(dis.data,cuImg.h_data,cuImg.width*cuImg.height*sizeof(float));
    Mat gray;
    dis.convertTo(gray,DataType<uchar>::type, 1, 0);

    cvNamedWindow("ss",CV_WINDOW_NORMAL);
    imshow("ss",gray);
    waitKey();
}
